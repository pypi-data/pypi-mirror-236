#include "hip/hip_runtime.h"
﻿#include "MD_core.cuh"
#include "thrust/sort.h"

#define BOX_TRAJ_COMMAND "box"
#define BOX_TRAJ_DEFAULT_FILENAME "mdbox.txt"
#define TRAJ_COMMAND "crd"
#define TRAJ_DEFAULT_FILENAME "mdcrd.dat"
#define RESTART_COMMAND "rst"
#define RESTART_DEFAULT_FILENAME "restart"
//20210827用于输出速度和力
#define FRC_TRAJ_COMMAND "frc"
#define VEL_TRAJ_COMMAND "vel"

//20230303 用于错误输出原因
#define ATOM_NUMBERS_DISMATCH "Reason:\n\t'atom_numbers' (the number of atoms) is diiferent in different input files\n"
#define ATOM_NUMBERS_MISSING "Reason:\n\tno 'atom_numbers' (the number of atoms) found. No 'mass_in_file' or 'amber_parm7' is provided\n"


//用来将原子的真实坐标转换为unsigned int坐标,注意factor需要乘以0.5（保证越界坐标自然映回box）
static __global__ void Crd_To_Uint_Crd(const int atom_numbers, const VECTOR box_length_inverse, const VECTOR *crd, UNSIGNED_INT_VECTOR *uint_crd)
{
    int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (atom_i < atom_numbers)
    {
        UNSIGNED_INT_VECTOR uinti;
        VECTOR temp = crd[atom_i];

        temp.x *= box_length_inverse.x;
        temp.y *= box_length_inverse.y;
        temp.z *= box_length_inverse.z;

        temp.x -= floorf(temp.x);
        temp.y -= floorf(temp.y);
        temp.z -= floorf(temp.z);

        uinti.uint_x = CONSTANT_UINT_MAX_FLOAT * temp.x;
        uinti.uint_y = CONSTANT_UINT_MAX_FLOAT * temp.y;
        uinti.uint_z = CONSTANT_UINT_MAX_FLOAT * temp.z;

        uint_crd[atom_i] = uinti;
    }
}

static __global__ void MD_Iteration_Leap_Frog
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        acc[i].x = inverse_mass[i] * frc[i].x;
        acc[i].y = inverse_mass[i] * frc[i].y;
        acc[i].z = inverse_mass[i] * frc[i].z;

        vel[i].x = vel[i].x + dt*acc[i].x;
        vel[i].y = vel[i].y + dt*acc[i].y;
        vel[i].z = vel[i].z + dt*acc[i].z;

        crd[i].x = crd[i].x + dt*vel[i].x;
        crd[i].y = crd[i].y + dt*vel[i].y;
        crd[i].z = crd[i].z + dt*vel[i].z;

    }
}

static __global__ void MD_Iteration_Leap_Frog_With_Max_Velocity
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt, const float max_velocity)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        VECTOR acc_i = inverse_mass[i] * frc[i];
        VECTOR vel_i = vel[i] + dt * acc_i;
        vel_i = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
        vel[i] = vel_i;
        crd[i] = crd[i] + dt * vel_i;
    }
}

static __global__ void MD_Iteration_Gradient_Descent
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
        crd[i] = crd[i] + dt * vel[i];
    }
}

static __global__ void MD_Iteration_Gradient_Descent_With_Max_Move
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep, float max_move)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
        VECTOR move = dt * vel[i];
        move = Make_Vector_Not_Exceed_Value(move, max_move);
        crd[i] = crd[i] + move;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_1(const int atom_numbers, const float half_dt, const float dt, const VECTOR *acc, VECTOR *vel, VECTOR *crd, VECTOR *frc)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        vel[i].x = vel[i].x + half_dt*acc[i].x;
        vel[i].y = vel[i].y + half_dt*acc[i].y;
        vel[i].z = vel[i].z + half_dt*acc[i].z;
        crd[i].x = crd[i].x + dt*vel[i].x;
        crd[i].y = crd[i].y + dt*vel[i].y;
        crd[i].z = crd[i].z + dt*vel[i].z;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_2(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        acc[i].x = inverse_mass[i] * frc[i].x;
        acc[i].y = inverse_mass[i] * frc[i].y;
        acc[i].z = inverse_mass[i] * frc[i].z;
        vel[i].x = vel[i].x + half_dt*acc[i].x;
        vel[i].y = vel[i].y + half_dt*acc[i].y;
        vel[i].z = vel[i].z + half_dt*acc[i].z;
    }
}

static __global__ void MD_Iteration_Speed_Verlet_2_With_Max_Velocity(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc, const float max_velocity)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < atom_numbers)
    {
        VECTOR acc_i = inverse_mass[i] * frc[i];
        VECTOR vel_i = vel[i] + half_dt * acc_i;

        vel[i] = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
        acc[i] = acc_i;
    }
}

static __global__ void Get_Origin(const int residue_numbers, const int *start, const int *end,
    const VECTOR *crd, const float *atom_mass, const float *residue_mass_inverse, VECTOR *center_of_mass)
{
    for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
    {
        VECTOR origin = { FLT_MAX, FLT_MAX, FLT_MAX };
        for (int atom_i = start[residue_i]; atom_i < end[residue_i]; atom_i += 1)
        {
            VECTOR crd_i = crd[atom_i]; 
            origin.x = fminf(origin.x, crd_i.x);
            origin.y = fminf(origin.y, crd_i.y);
            origin.z = fminf(origin.z, crd_i.z);
        }
        center_of_mass[residue_i] = origin;
    }
}

static __global__ void Get_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
    const VECTOR *crd, const float *atom_mass, const float *residue_mass_inverse, VECTOR *center_of_mass)
{
    for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
    {
        VECTOR com_lin = { 0.0f, 0.0f, 0.0f };
        for (int atom_i = start[residue_i]; atom_i < end[residue_i]; atom_i += 1)
        {
            com_lin = com_lin + atom_mass[atom_i] * crd[atom_i];
        }
        center_of_mass[residue_i] = residue_mass_inverse[residue_i] * com_lin;
    }
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
    const float scaler, const VECTOR *center_of_mass, const VECTOR box_length, const VECTOR *no_wrap_crd, VECTOR *crd)
{
    VECTOR trans_vec;
    VECTOR com;
    for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
    {
        com = center_of_mass[residue_i];

        trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
        trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
        trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
        trans_vec = scaler * trans_vec - com;

        for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
        {
            crd[atom_i] = no_wrap_crd[atom_i] + trans_vec;
        }
    }
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
    const VECTOR scaler, const VECTOR *center_of_mass, const VECTOR box_length, const VECTOR *no_wrap_crd, VECTOR *crd)
{
    VECTOR trans_vec;
    VECTOR com;
    for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
    {
        com = center_of_mass[residue_i];

        trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
        trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
        trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
        trans_vec.x = scaler.x * trans_vec.x - com.x;
        trans_vec.y = scaler.y * trans_vec.y - com.y;
        trans_vec.z = scaler.z * trans_vec.z - com.z;

        for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
        {
            crd[atom_i] = no_wrap_crd[atom_i] + trans_vec;
        }
    }
}

static __global__ void Add_Sum_List(int n, float *atom_virial, float *sum_virial)
{
    float temp = 0;
    for (int i = threadIdx.x; i < n; i = i + blockDim.x)
    {
        temp = temp + atom_virial[i];
    }
    atomicAdd(sum_virial, temp);
}

static __global__ void Calculate_Pressure_Cuda(const float V_inverse, const float *ek, const float *virial, float *pressure)
{
    pressure[0] = (ek[0] * 2 + virial[0]) * 0.33333333333333f * V_inverse;
}

static __global__ void MD_Residue_Ek
(const int residue_numbers,const int *start,const int *end,float *ek,
const VECTOR *atom_vel,const float *atom_mass)
{
    int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (residue_i < residue_numbers)
    {
        VECTOR momentum = {0.,0.,0.};
        float res_mass = 0.; //待提出，只需要初始时计算一遍
        int s = start[residue_i];
        int e = end[residue_i];
        float mass_lin;
        for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
        {
            mass_lin = atom_mass[atom_i];

            momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
            momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
            momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
            res_mass = res_mass + mass_lin;
        }
        ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass;
    }
}

static __global__ void MD_Atom_Ek
(const int atom_numbers, float *ek, const VECTOR *atom_vel, const float *atom_mass)
{
    int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
    if (atom_i < atom_numbers)
    {
        VECTOR v = atom_vel[atom_i];
        ek[atom_i] = 0.5 * v * v * atom_mass[atom_i];
    }
}

void MD_INFORMATION::system_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    steps = 0;
    if (md_info->mode != md_info->RERUN)
    {
        step_limit = 1000;
        if (controller[0].Command_Exist("step_limit"))
        {
            controller->Check_Int("step_limit", "MD_INFORMATION::system_information::Initial");
            step_limit = atoi(controller[0].Command("step_limit"));
        }

        target_temperature = 300.0f;
        if (md_info->mode >= md_info->NVT && controller[0].Command_Exist("target_temperature"))
        {
            controller->Check_Float("target_temperature", "MD_INFORMATION::system_information::Initial");
            target_temperature = atof(controller[0].Command("target_temperature"));
        }

        target_pressure = 1;
        if (md_info->mode == md_info->NPT && controller[0].Command_Exist("target_pressure"))
        {
            controller->Check_Float("target_pressure", "MD_INFORMATION::system_information::Initial");
            target_pressure = atof(controller[0].Command("target_pressure"));
        }
        target_pressure *= CONSTANT_PRES_CONVERTION_INVERSE;
    }
    else
    {
        step_limit = INT_MAX - 1;
        if (controller[0].Command_Exist("frame_limit"))
        {
            controller->Check_Int("frame_limit", "MD_INFORMATION::system_information::Initial");
            step_limit = atoi(controller[0].Command("frame_limit"));
        }
    }
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        std::set<int> temp;
        connectivity[i] = temp;
    }
    if (md_info->mode != md_info->RERUN)
    {
        controller->Step_Print_Initial("step", "%d");
        controller->Step_Print_Initial("time", "%.3lf");
        controller->Step_Print_Initial("temperature", "%.2f");
    }
    else
    {
        controller->Step_Print_Initial("frame", "%d");
    }
    Cuda_Malloc_Safely((void**)&this->d_virial, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_pressure, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_temperature, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_potential, sizeof(float));
    Cuda_Malloc_Safely((void**)&this->d_sum_of_atom_ek, sizeof(float));
}

void MD_INFORMATION::non_bond_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    if (controller[0].Command_Exist("skin"))
    {
        controller->Check_Float("skin", "MD_INFORMATION::non_bond_information::Initial");
        skin = atof(controller[0].Command("skin"));
    }
    else
    {
        skin = 2.0;
    }
    controller->printf("    skin set to %.2f Angstrom\n", skin);

    if (controller[0].Command_Exist("cutoff"))
    {
        controller->Check_Float("cutoff", "MD_INFORMATION::non_bond_information::Initial");
        cutoff = atof(controller[0].Command("cutoff"));
    }
    else
    {
        cutoff = 10.0;
    }
    controller->printf("    cutoff set to %.2f Angstrom\n", cutoff);
    /*===========================
    读取排除表相关信息
    ============================*/
    if (controller[0].Command_Exist("exclude_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading excluded list:\n");
        Open_File_Safely(&fp, controller[0].Command("exclude_in_file"), "r");
        
        int atom_numbers = 0;
        int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &excluded_atom_numbers);
        if (scanf_ret != 2)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of exclude_in_file is not right\n");
        }
        if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::non_bond_information::Initial", ATOM_NUMBERS_DISMATCH);
        }
        else if (md_info->atom_numbers == 0)
        {
            md_info->atom_numbers = atom_numbers;
        }
        controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

        Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

        Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
        int count = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%d", &h_excluded_numbers[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "Reason:\n\tThe format of exclude_in_file is not right\n");
            }
            h_excluded_list_start[i] = count;
            for (int j = 0; j < h_excluded_numbers[i]; j++)
            {
                scanf_ret = fscanf(fp, "%d", &h_excluded_list[count]);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "Reason:\n\tThe format of exclude_in_file is not right\n");
                }
                count++;
            }
        }
        if (count != excluded_atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", 
                "Reason:\n\tThe format of exclude_in_file is not right (excluded_atom_numbers is not right)\n");
        }
        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading excluded list\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        /*===========================
        从parm中读取排除表相关信息
        ============================*/
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller->printf("    Start reading excluded list from AMBER file:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (!fgets(temps, CHAR_LENGTH_MAX, parm))
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d\n", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::non_bond_information::Initial", ATOM_NUMBERS_DISMATCH);
                }
                else if (md_info->atom_numbers == 0)
                {
                    md_info->atom_numbers = atom_numbers;
                }
                Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
                Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);

                Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
                Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
                for (int i = 0; i < 9; i = i + 1)
                {
                    scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
                scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

                Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);
                Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
            }

            //read atom_excluded_number for every atom
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "NUMBER_EXCLUDED_ATOMS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i<md_info->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%d\n", &h_excluded_numbers[i]);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
            }
            //read every atom's excluded atom list
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "EXCLUDED_ATOMS_LIST") == 0)
            {
                int count = 0;
                int lin = 0;
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i<md_info->atom_numbers; i = i + 1)
                {
                    h_excluded_list_start[i] = count;
                    for (int j = 0; j<h_excluded_numbers[i]; j = j + 1)
                    {
                        int scanf_ret = fscanf(parm, "%d\n", &lin);
                        if (scanf_ret != 1)
                        {
                            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                        }
                        if (lin == 0)
                        {
                            h_excluded_numbers[i] = 0;
                            break;
                        }
                        else
                        {
                            h_excluded_list[count] = lin - 1;
                            count = count + 1;
                        }
                    }
                    if (h_excluded_numbers[i] > 0)
                        thrust::sort(&h_excluded_list[h_excluded_list_start[i]], &h_excluded_list[h_excluded_list_start[i]] + h_excluded_numbers[i]);
                }
            }
        }

        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading excluded list from AMBER file\n\n");
        fclose(parm);
    }
    else
    {
        int atom_numbers = md_info->atom_numbers;
        excluded_atom_numbers = 0;
        controller->printf("    Set all atom exclude no atoms as default\n"); 

        Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
        Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

        Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
        Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);


        int count = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            h_excluded_numbers[i] = 0;
            h_excluded_list_start[i] = count;
            for (int j = 0; j < h_excluded_numbers[i]; j++)
            {
                h_excluded_list[count] = 0;
                count++;
            }
        }
        hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
    }

}

void MD_INFORMATION::periodic_box_condition_information::Initial(CONTROLLER *controller, MD_INFORMATION* md_info)
{
    this->md_info = md_info;
    this->uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * md_info->sys.box_length;
    this->md_info->MD_Information_Crd_To_Uint_Crd();
}

void MD_INFORMATION::Read_Mode(CONTROLLER *controller)
{
    if (controller->Command_Choice("mode", "nvt"))
    {
        controller->printf("    Mode set to NVT\n");
        mode = 1;
    }
    else if (controller->Command_Choice("mode", "npt"))
    {
        controller->printf("    Mode set to NPT\n");
        mode = 2;
    }
    else if (controller->Command_Choice("mode", "minimization"))
    {
        controller->printf("    Mode set to Energy Minimization\n");
        mode = -1;
    }
    else if (controller->Command_Choice("mode", "nve"))
    {
        controller->printf("    Mode set to NVE\n");
        mode = 0;
    }
    else if (controller->Command_Choice("mode", "rerun"))
    {
        controller->printf("    Mode set to RERUN\n");
        mode = -2;
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Mode", "Reason:\n\t'mode' is required\n");
    }
}

void MD_INFORMATION::Read_dt(CONTROLLER *controller)
{
    if (mode == RERUN)
    {
        return;
    }
    if (controller[0].Command_Exist("dt"))
    {
        controller->Check_Float("dt", "MD_INFORMATION::Read_dt");
        controller->printf("    dt set to %f ps\n", atof(controller[0].Command("dt")));
        dt = atof(controller[0].Command("dt")) * CONSTANT_TIME_CONVERTION;
        sscanf(controller[0].Command("dt"), "%lf", &sys.dt_in_ps);
    }
    else
    {
        if (mode != MINIMIZATION)
            dt = 0.001;
        else
            dt = 1e-8;
        sys.dt_in_ps = 0.001;
        controller->printf("    dt set to %e ps\n", dt);
        dt *= CONSTANT_TIME_CONVERTION;
    }
    if (mode == MINIMIZATION)
    {
        sys.dt_in_ps = 0;
    }
}

void MD_INFORMATION::trajectory_output::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    current_crd_synchronized_step = -1;
    if (md_info->mode != md_info->RERUN)
    {
        int default_interval = 1000;
        if (controller[0].Command_Exist("write_information_interval"))
        {
            controller->Check_Int("write_information_interval", "MD_INFORMATION::trajectory_output::Initial");
            default_interval = atoi(controller[0].Command("write_information_interval"));
        }
        write_trajectory_interval = default_interval;        
        if (controller[0].Command_Exist("write_trajectory_interval"))
        {
            controller->Check_Int("write_trajectory_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_trajectory_interval = atoi(controller[0].Command("write_trajectory_interval"));
        }
        write_mdout_interval = default_interval;
        if (controller[0].Command_Exist("write_mdout_interval"))
        {
            controller->Check_Int("write_mdout_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_mdout_interval = atoi(controller[0].Command("write_mdout_interval"));
        }
        write_restart_file_interval = default_interval;
        if (controller[0].Command_Exist("write_restart_file_interval"))
        {
            controller->Check_Int("write_restart_file_interval", "MD_INFORMATION::trajectory_output::Initial");
            write_restart_file_interval = atoi(controller[0].Command("write_restart_file_interval"));
        }
        if (controller->Command_Exist(RESTART_COMMAND))
        {
            strcpy(restart_name, controller->Command(RESTART_COMMAND));
        }
        else
        {
            strcpy(restart_name, RESTART_DEFAULT_FILENAME);
        }
        if (controller->Command_Exist(FRC_TRAJ_COMMAND))
        {
            is_frc_traj = 1;
            Open_File_Safely(&frc_traj, controller->Command(FRC_TRAJ_COMMAND), "wb");
        }
        if (controller->Command_Exist(VEL_TRAJ_COMMAND))
        {
            is_vel_traj = 1;
            Open_File_Safely(&vel_traj, controller->Command(VEL_TRAJ_COMMAND), "wb");
        }
    }
    else
    {
        write_trajectory_interval = 0;
        write_mdout_interval = 1;
        write_restart_file_interval = 0;
    }

    if (write_trajectory_interval != 0)
    {
        if (controller->Command_Exist(TRAJ_COMMAND))
        {
            Open_File_Safely(&crd_traj, controller->Command(TRAJ_COMMAND), "wb");
        }
        else
        {
            Open_File_Safely(&crd_traj, TRAJ_DEFAULT_FILENAME, "wb");
        }
        if (controller->Command_Exist(BOX_TRAJ_COMMAND))
        {
            Open_File_Safely(&box_traj, controller->Command(BOX_TRAJ_COMMAND), "w");
        }
        else
        {
            Open_File_Safely(&box_traj, BOX_TRAJ_DEFAULT_FILENAME, "w");
        }
    }
}

void MD_INFORMATION::NVE_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    max_velocity = -1;
    if (controller[0].Command_Exist("velocity_max"))
    {
        controller->Check_Float("velocity_max", "MD_INFORMATION::NVE_iteration::Initial");
        max_velocity = atof(controller[0].Command("velocity_max"));
    }
}
void MD_INFORMATION::residue_information::Read_AMBER_Parm7(const char *file_name, CONTROLLER controller)
{
    FILE *parm = NULL;
    Open_File_Safely(&parm, file_name, "r");
    controller.printf("    Start reading residue informataion from AMBER parm7:\n");
    
    while (true)
    {
        char temps[CHAR_LENGTH_MAX];
        char temp_first_str[CHAR_LENGTH_MAX];
        char temp_second_str[CHAR_LENGTH_MAX];
        if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
        {
            break;
        }
        if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
        {
            continue;
        }
        //read in atomnumber atomljtypenumber
        if (strcmp(temp_first_str, "%FLAG") == 0
            && strcmp(temp_second_str, "POINTERS") == 0)
        {
            char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

            int atom_numbers = 0;
            int scanf_ret = fscanf(parm, "%d", &atom_numbers);
            if (scanf_ret != 1)
            {
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
            }
            if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
            {
                controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", ATOM_NUMBERS_DISMATCH);
            }
            else if (md_info->atom_numbers == 0)
            {
                md_info->atom_numbers = atom_numbers;
            }
            for (int i = 0; i < 10; i = i + 1)
            {
                int lin;
                scanf_ret = fscanf(parm, "%d\n", &lin);
                if (scanf_ret != 1)
                {
                    controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
                }
            }
            scanf_ret = fscanf(parm, "%d\n", &this->residue_numbers);//NRES
            if (scanf_ret != 1)
            {
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
            }
            controller.printf("        residue_numbers is %d\n", this->residue_numbers);

            Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

            Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers); 
            Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
        }//FLAG POINTERS

        //residue range read
        if (strcmp(temp_first_str, "%FLAG") == 0
            && strcmp(temp_second_str, "RESIDUE_POINTER") == 0)
        {
            char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
            //注意读进来的数的编号要减1
            int *lin_serial;
            Malloc_Safely((void**)&lin_serial, sizeof(int)* this->residue_numbers);
            for (int i = 0; i<this->residue_numbers; i = i + 1)
            {
                int scanf_ret = fscanf(parm, "%d\n", &lin_serial[i]);
                if (scanf_ret != 1)
                {
                    controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Read_AMBER_Parm7", "Reason:\n\tthe format of the amber_parm7 is not right\n");
                }
            }
            for (int i = 0; i<this->residue_numbers - 1; i = i + 1)
            {
                h_res_start[i] = lin_serial[i] - 1;
                h_res_end[i] = lin_serial[i + 1] - 1;
            }
            h_res_start[this->residue_numbers - 1] = lin_serial[this->residue_numbers - 1] - 1;
            h_res_end[this->residue_numbers - 1] = md_info->atom_numbers + 1 - 1;

            free(lin_serial);
        }
    }//while cycle

    hipMemcpy(this->d_res_start, h_res_start, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);
    hipMemcpy(this->d_res_end, h_res_end, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);

    controller.printf("    End reading residue informataion from AMBER parm7\n\n");

    fclose(parm);
}

void MD_INFORMATION::residue_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (!(controller[0].Command_Exist("residue_in_file")))
    {
        if (controller[0].Command_Exist("amber_parm7"))
        {
            Read_AMBER_Parm7(controller[0].Command("amber_parm7"), controller[0]);
            is_initialized = 1;
        }
        //对于没有residue输入的模拟，默认每个粒子作为一个residue
        else
        {
            residue_numbers = md_info->atom_numbers;
            controller->printf("    Set default residue list:\n");
            controller->printf("        residue_numbers is %d\n", residue_numbers);
            Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
            Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
            Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

            Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
            Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
            int count = 0;
            int temp=1;//每个粒子作为一个residue
            for (int i = 0; i < residue_numbers; i++)
            {
                h_res_start[i] = count;
                count += temp;
                h_res_end[i] = count;
            }
            hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
            hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
            controller->printf("    End reading residue list\n\n");
            is_initialized = 1;
        }
    }
    else
    {
        FILE *fp = NULL;
        controller->printf("    Start reading residue list:\n");
        Open_File_Safely(&fp, controller[0].Command("residue_in_file"), "r");
        int atom_numbers = 0;
        int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &residue_numbers);
        if (scanf_ret != 2)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Initial", "Reason:\n\tthe format of the residue_in_file is not right\n");
        }
        if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::residue_information::Initial", ATOM_NUMBERS_DISMATCH);
        }
        else if (md_info->atom_numbers == 0)
        {
            md_info->atom_numbers = atom_numbers;
        }
        controller->printf("        residue_numbers is %d\n", residue_numbers);
        Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
        Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
        Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
        Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
        Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
        Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));

        int count = 0;
        int temp;
        for (int i = 0; i < residue_numbers; i++)
        {
            h_res_start[i] = count;
            scanf_ret = fscanf(fp, "%d", &temp);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::residue_information::Initial", "Reason:\n\tthe format of the residue_in_file is not right\n");
            }
            count += temp;
            h_res_end[i] = count;
        }
        hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
        controller->printf("    End reading residue list\n\n");
        fclose(fp);
        is_initialized = 1;
    }
    if (is_initialized)
    {
        if (md_info->h_mass != NULL)
        {
            for (int i = 0; i < residue_numbers; i++)
            {
                float temp_mass = 0;
                for (int j = h_res_start[i]; j < h_res_end[i]; j++)
                {
                    temp_mass += md_info->h_mass[j];
                }
                this->h_mass[i] = temp_mass;
                if (temp_mass == 0)
                    this->h_mass_inverse[i] = 0;
                else
                    this->h_mass_inverse[i] = 1.0 / temp_mass;
            }
            hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
            hipMemcpy(d_mass, h_mass, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
        }
        else
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
                                           "MD_INFORMATION::residue_information::Initial",
                                           "Reason:\n\tno mass information found");
        }
    }
}

void MD_INFORMATION::Read_Coordinate_And_Velocity(CONTROLLER *controller)
{
    sys.start_time = 0.0;
    if (mode == RERUN)
    {
        if (atom_numbers == 0)
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Coordinate_And_Velocity", 
                "Reason:\n\tFor the 'rerun' mode, the number of atoms should be provided by mass_in_file or charge_in_file\n");
        }
        this->atom_numbers = atom_numbers;
        Malloc_Safely((void**)&coordinate, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));            
        Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
        sys.box_length = { 100.0f, 100.0f, 100.0f };
        rerun.Initial(controller, this);
        rerun.Iteration(rerun.start_frame);
        sys.box_length.x *= rerun.box_length_change_factor.x;
        sys.box_length.y *= rerun.box_length_change_factor.y;
        sys.box_length.z *= rerun.box_length_change_factor.z;
    }
    else if (controller[0].Command_Exist("coordinate_in_file"))
    {
        Read_Coordinate_In_File(controller[0].Command("coordinate_in_file"), controller[0]);
        if (controller[0].Command_Exist("velocity_in_file"))
        {
            FILE *fp = NULL;
            controller->printf("    Start reading velocity_in_file:\n");
            Open_File_Safely(&fp, controller[0].Command("velocity_in_file"), "r");
            
            int atom_numbers = 0;
            char lin[CHAR_LENGTH_MAX];
            char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
            int scanf_ret = sscanf(lin, "%d", &atom_numbers);
            if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
            {
                controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Coordinate_And_Velocity", ATOM_NUMBERS_DISMATCH);
            }
            Malloc_Safely((void**)&velocity, sizeof(VECTOR)* (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)* (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            for (int i = 0; i < atom_numbers; i++)
            {
                scanf_ret = fscanf(fp, "%f %f %f", &velocity[i].x, &velocity[i].y, &velocity[i].z);
                if (scanf_ret != 3)
                {
                    std::string error_reason = "Reason:\n\tthe format of the velocity_in_file (";
                    error_reason += controller->Command("velocity_in_file");
                    error_reason += ") is not right (missing the velocity of atom ";
                    error_reason += i;
                    error_reason += ")\n";
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_And_Velocity", error_reason.c_str());
                }
            }
            hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
            controller->printf("    End reading velocity_in_file\n\n");
            fclose(fp);
        }
        else
        {
            controller->printf("    Velocity is set to zero as default\n");
            Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
            for (int i = 0; i < atom_numbers; i++)
            {
                velocity[i].x = 0;
                velocity[i].y = 0;
                velocity[i].z = 0;
            }
            hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
        }
    }
    else if (controller[0].Command_Exist("amber_rst7"))
    {
        output.amber_irest = 1;
        if (controller[0].Command_Exist("amber_irest"))
        {
            output.amber_irest = controller->Get_Bool("amber_irest", "MD_INFORMATION::Read_Coordinate_And_Velocity");
        }
        Read_Rst7(controller[0].Command("amber_rst7"), output.amber_irest, controller[0]);
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
            "MD_INFORMATION::Read_Coordinate_And_Velocity",
            "Reason:\n\tno coordinate information found");
    }
}

void MD_INFORMATION::Read_Mass(CONTROLLER *controller)
{
    if (controller[0].Command_Exist("mass_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading mass:\n");
        Open_File_Safely(&fp, controller[0].Command("mass_in_file"), "r");
        int atom_numbers = 0;
        char lin[CHAR_LENGTH_MAX];
        char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
        int scanf_ret = sscanf(lin, "%d", &atom_numbers);
        if (scanf_ret != 1)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of mass_in_file is not right\n");
        }
        if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_DISMATCH);
        }
        else if (this->atom_numbers == 0)
        {
            this->atom_numbers = atom_numbers;
        }
        Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
        sys.total_mass = 0;
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%f", &h_mass[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of mass_in_file is not right\n");
            }
            sys.total_mass += h_mass[i];
            if (h_mass[i] == 0)
                h_mass_inverse[i] = 0;
            else
                h_mass_inverse[i] = 1.0 / h_mass[i];

        }
        controller->printf("    End reading mass\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller[0].printf("    Start reading mass from AMBER parm7:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            //read in atomnumber atomljtypenumber
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_DISMATCH);
                }
                else if (this->atom_numbers == 0)
                {
                    this->atom_numbers = atom_numbers;
                }
                Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
                Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "MASS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                double lin;
                sys.total_mass = 0;
                for (int i = 0; i < this->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%lf\n", &lin);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                    this->h_mass[i] = (float)lin;
                    if (h_mass[i] == 0)
                        h_mass_inverse[i] = 0;
                    else
                        h_mass_inverse[i] = 1.0f / h_mass[i];
                    sys.total_mass += h_mass[i];
                }
            }
        }
        controller[0].printf("    End reading mass from AMBER parm7\n\n");
        fclose(parm);
    }
    else if (atom_numbers > 0)
    {
        controller[0].printf("    mass is set to 20 as default\n");
        sys.total_mass = 0;
        Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
        Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            h_mass[i] = 20;
            h_mass_inverse[i] = 1.0 / h_mass[i];
            sys.total_mass += h_mass[i];
        }
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Mass", ATOM_NUMBERS_MISSING);
    }
    if (atom_numbers > 0)
    {
        hipMemcpy(d_mass, h_mass, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
        hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
    }
}

void MD_INFORMATION::Read_Charge(CONTROLLER *controller)
{
    if (controller[0].Command_Exist("charge_in_file"))
    {
        FILE *fp = NULL;
        controller->printf("    Start reading charge:\n");
        Open_File_Safely(&fp, controller[0].Command("charge_in_file"), "r");
        int atom_numbers = 0;
        char lin[CHAR_LENGTH_MAX];
        char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
        int scanf_ret = sscanf(lin, "%d", &atom_numbers);
        if (scanf_ret != 1)
        {
            controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of charge_in_file is not right\n");
        }
        if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_DISMATCH);
        }
        else if (this->atom_numbers == 0)
        {
            this->atom_numbers = atom_numbers;
        }
        Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            scanf_ret = fscanf(fp, "%f", &h_charge[i]);
            if (scanf_ret != 1)
            {
                controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of charge_in_file is not right\n");
            }
        }
        controller->printf("    End reading charge\n\n");
        fclose(fp);
    }
    else if (controller[0].Command_Exist("amber_parm7"))
    {
        FILE *parm = NULL;
        Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
        controller[0].printf("    Start reading charge from AMBER parm7:\n");
        while (true)
        {
            char temps[CHAR_LENGTH_MAX];
            char temp_first_str[CHAR_LENGTH_MAX];
            char temp_second_str[CHAR_LENGTH_MAX];
            if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
            {
                break;
            }
            if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
            {
                continue;
            }
            //read in atomnumber atomljtypenumber
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "POINTERS") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

                int atom_numbers = 0;
                int scanf_ret = fscanf(parm, "%d", &atom_numbers);
                if (scanf_ret != 1)
                {
                    controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                }
                if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
                {
                    controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_DISMATCH);
                }
                else if (this->atom_numbers == 0)
                {
                    this->atom_numbers = atom_numbers;
                }
                Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
                Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
            }
            if (strcmp(temp_first_str, "%FLAG") == 0
                && strcmp(temp_second_str, "CHARGE") == 0)
            {
                char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
                for (int i = 0; i < this->atom_numbers; i = i + 1)
                {
                    int scanf_ret = fscanf(parm, "%f", &h_charge[i]);
                    if (scanf_ret != 1)
                    {
                        controller->Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::non_bond_information::Initial", "The format of amber_parm7 is not right\n");
                    }
                }
            }
        }
        controller[0].printf("    End reading charge from AMBER parm7\n\n");
        fclose(parm);
    }
    else if (atom_numbers > 0)
    {
        controller[0].printf("    charge is set to 0 as default\n");
        Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
        Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
        for (int i = 0; i < atom_numbers; i++)
        {
            h_charge[i] = 0;
        }
    }
    else
    {
        controller->Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::Read_Charge", ATOM_NUMBERS_MISSING);
    }
    if (atom_numbers > 0)
    {
        hipMemcpy(d_charge, h_charge, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
    }
}

//MD_INFORMATION成员函数
void MD_INFORMATION::Initial(CONTROLLER *controller)
{
    controller->printf("START INITIALIZING MD CORE:\n");
    atom_numbers = 0; //初始化，使得能够进行所有原子数目是否相等的判断
    
    strcpy(md_name, controller[0].Command("md_name"));
    Read_Mode(controller);
    Read_dt(controller);

    Read_Mass(controller);
    Read_Charge(controller);
    Atom_Information_Initial();

    Read_Coordinate_And_Velocity(controller);

    sys.Initial(controller, this);
    nb.Initial(controller, this);
    
    output.Initial(controller, this);

    nve.Initial(controller, this);
    
    min.Initial(controller, this);

    res.Initial(controller, this);

    mol.Initial(controller, this);

    pbc.Initial(controller, this);
    
    is_initialized = 1;
    controller->printf("    structure last modify date is %d\n", last_modify_date);
    controller->printf("END INITIALIZING MD CORE\n\n");
}

void MD_INFORMATION::Atom_Information_Initial()
{
    Malloc_Safely((void**)&this->force, sizeof(VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Malloc_Safely((void**)&this->h_atom_energy, sizeof(float) * atom_numbers);
    Malloc_Safely((void**)&this->h_atom_virial, sizeof(double)* atom_numbers);
    Cuda_Malloc_Safely((void**)&this->acc, sizeof(VECTOR) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->frc, sizeof(VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&this->uint_crd, sizeof(UNSIGNED_INT_VECTOR) * (atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&this->d_atom_energy, sizeof(float) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->d_atom_virial, sizeof(float) * atom_numbers);
    Cuda_Malloc_Safely((void**)&this->d_atom_ek, sizeof(float) * atom_numbers);
    Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
        (3 * this->atom_numbers, (float*)this->acc, 0.);
    Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
        (3 * this->atom_numbers, (float*)this->frc, 0.);
    sys.freedom = 3 * atom_numbers; //最大自由度，后面减
}

void MD_INFORMATION::Read_Coordinate_In_File(const char* file_name, CONTROLLER controller)
{
    FILE *fp =NULL;
    controller.printf("    Start reading coordinate_in_file:\n");
    Open_File_Safely(&fp, file_name, "r");
    char lin[CHAR_LENGTH_MAX];
    char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
    int atom_numbers = 0;
    int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
    if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
    {
        controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Coordinate_In_File", ATOM_NUMBERS_DISMATCH);
    }
    else if (this->atom_numbers == 0)
    {
        this->atom_numbers = atom_numbers;
    }
    if (scanf_ret == 0)
    {
        std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
    }
    else  if (scanf_ret == 1)
    {
        sys.start_time = 0;
    }

    controller.printf("        atom_numbers is %d\n", this->atom_numbers);
    controller.printf("        system start_time is %lf\n", this->sys.start_time);
    Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));

    for (int i = 0; i < atom_numbers; i++)
    {
        scanf_ret = fscanf(fp, "%f %f %f", &coordinate[i].x, &coordinate[i].y, &coordinate[i].z);
        if (scanf_ret != 3)
        {
            std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
            error_reason += file_name;
            error_reason += ") is not right\n";
            controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
        }
    }
    scanf_ret = fscanf(fp, "%f %f %f", &sys.box_length.x, &sys.box_length.y, &sys.box_length.z);
    if (scanf_ret != 3)
    {
        std::string error_reason = "Reason:\n\tthe format of the coordinate_in_file (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Coordinate_In_File", error_reason.c_str());
    }
    controller.printf("        box_length is\n            x: %f\n            y: %f\n            z: %f\n", sys.box_length.x, sys.box_length.y, sys.box_length.z);
    hipMemcpy(crd, coordinate, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
    controller.printf("    End reading coordinate_in_file\n\n");
    fclose(fp);
}
void MD_INFORMATION::Read_Rst7(const char* file_name, int irest, CONTROLLER controller)
{
    FILE *fin = NULL;
    Open_File_Safely(&fin, file_name, "r");
    controller.printf("    Start reading AMBER rst7:\n");
    char lin[CHAR_LENGTH_MAX];
    int atom_numbers = 0;
    char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
    get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
    int has_vel = 0;
    int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
    if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
    {
        controller.Throw_SPONGE_Error(spongeErrorConflictingCommand, "MD_INFORMATION::Read_Rst7", ATOM_NUMBERS_DISMATCH);
    }
    else if (this->atom_numbers == 0)
    {
        this->atom_numbers = atom_numbers;
    }
    if (scanf_ret == 0)
    {
        std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
        error_reason += file_name;
        error_reason += ") is not right\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
    }
    else  if (scanf_ret == 2)
    {
        has_vel = 1;
    }
    else
    {
        sys.start_time = 0;
    }

    Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*(this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Malloc_Safely((void**)&velocity, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));
    Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * (this->atom_numbers + no_direct_interaction_virtual_atom_numbers));

    controller.printf("        atom_numbers is %d\n", this->atom_numbers);
    controller.printf("        system start time is %lf\n", this->sys.start_time);

    if (has_vel == 0 || irest == 0)
    {
        controller.printf("        All velocity will be set to 0\n");
    }


    for (int i = 0; i < this->atom_numbers; i = i + 1)
    {
        scanf_ret = fscanf(fin, "%f %f %f",
            &this->coordinate[i].x,
            &this->coordinate[i].y,
            &this->coordinate[i].z);
        if (scanf_ret != 3)
        {
            std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
            error_reason += file_name;
            error_reason += ") is not right (missing the coordinate of atom ";
            error_reason += i;
            error_reason += ")\n";
            controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
        }
    }
    if (has_vel)
    {
        for (int i = 0; i < this->atom_numbers; i = i + 1)
        {
            scanf_ret = fscanf(fin, "%f %f %f",
                &this->velocity[i].x,
                &this->velocity[i].y,
                &this->velocity[i].z);
            if (scanf_ret != 3)
            {
                std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
                error_reason += file_name;
                error_reason += ") is not right (missing the velocity of atom ";
                error_reason += i;
                error_reason += ")\n";
                controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
            }
        }
    }
    if (irest == 0 || !has_vel)
    {
        for (int i = 0; i < this->atom_numbers; i = i + 1)
        {
            this->velocity[i].x = 0.0;
            this->velocity[i].y = 0.0;
            this->velocity[i].z = 0.0;
        }
    }
    scanf_ret = fscanf(fin, "%f %f %f", &this->sys.box_length.x, &this->sys.box_length.y, &this->sys.box_length.z);
    if (scanf_ret != 3)
    {
        std::string error_reason = "Reason:\n\tthe format of the amber_rst7 (";
        error_reason += file_name;
        error_reason += ") is not right (missing the box information)\n";
        controller.Throw_SPONGE_Error(spongeErrorBadFileFormat, "MD_INFORMATION::Read_Rst7", error_reason.c_str());
    }
    controller.printf("        system size is %f %f %f\n", this->sys.box_length.x, this->sys.box_length.y, this->sys.box_length.z);
    hipMemcpy(this->crd, this->coordinate, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
    hipMemcpy(this->vel, this->velocity, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
    fclose(fin);
    controller.printf("    End reading AMBER rst7\n\n");
}

void MD_INFORMATION::trajectory_output::Append_Crd_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        md_info->Crd_Vel_Device_To_Host();
        if (fp == NULL)
        {
            fp = crd_traj;
        }
        fwrite(&md_info->coordinate[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
    }
}

// 20210827用于输出速度和力
void MD_INFORMATION::trajectory_output::Append_Frc_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        hipMemcpy(md_info->force, md_info->frc, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
        if (fp == NULL)//默认的frc输出位置
        {
            fp = frc_traj;
            if (fp != NULL)
            {
                fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
            }
        }
        else
        {
            fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
        }
    }
}
void MD_INFORMATION::trajectory_output::Append_Vel_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        hipMemcpy(md_info->velocity, md_info->vel, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
        if (fp == NULL)//默认的vel输出位置
        {
            fp = vel_traj;
            if (fp != NULL)
            {
                fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
            }
        }
        else
        {
            fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
        }
    }
}

void MD_INFORMATION::trajectory_output::Append_Box_Traj_File(FILE *fp)
{
    if (md_info->is_initialized)
    {
        if (fp == NULL)
        {
            fp = box_traj;
        }
        fprintf(fp, "%f %f %f %.0f %.0f %.0f\n", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
    }
}

void MD_INFORMATION::trajectory_output::Export_Restart_File(const char *rst7_name)
{
    if (!md_info->is_initialized)
        return;

    char filename[CHAR_LENGTH_MAX];
    if (rst7_name == NULL)
        strcpy(filename, restart_name);
    else
        strcpy(filename, rst7_name);
    md_info->Crd_Vel_Device_To_Host();
    if (amber_irest >= 0)
    {
        const char *sys_name = md_info->md_name;
        FILE *lin = NULL;
        Open_File_Safely(&lin, filename, "w");
        fprintf(lin, "%s\n", sys_name);
        fprintf(lin, "%8d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        int s = 0;
        for (int i = 0; i < md_info->atom_numbers; i = i + 1)
        {
            fprintf(lin, "%12.7f%12.7f%12.7f", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
            s = s + 1;
            if (s == 2)
            {
                s = 0;
                fprintf(lin, "\n");
            }
        }
        if (s == 1)
        {
            s = 0;
            fprintf(lin, "\n");
        }
        for (int i = 0; i < md_info->atom_numbers; i = i + 1)
        {
            fprintf(lin, "%12.7f%12.7f%12.7f", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
            s = s + 1;
            if (s == 2)
            {
                s = 0;
                fprintf(lin, "\n");
            }
        }
        if (s == 1)
        {
            s = 0;
            fprintf(lin, "\n");
        }
        fprintf(lin, "%12.7f%12.7f%12.7f", (float)md_info->sys.box_length.x, (float)md_info->sys.box_length.y, (float)md_info->sys.box_length.z);
        fprintf(lin, "%12.7f%12.7f%12.7f", (float)90., (float)90., (float)90.);
        fclose(lin);
    }
    else
    {
        FILE *lin = NULL;
        FILE *lin2 = NULL;
        std::string buffer;
        buffer = std::string(filename) + "_coordinate.txt";
        Open_File_Safely(&lin, buffer.c_str(), "w");
        buffer = std::string(filename) + "_velocity.txt";
        Open_File_Safely(&lin2, buffer.c_str(), "w");
        fprintf(lin, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        fprintf(lin2, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
        for (int i = 0; i < md_info->atom_numbers; i++)
        {
            fprintf(lin, "%12.7f %12.7f %12.7f\n", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
            fprintf(lin2, "%12.7f %12.7f %12.7f\n", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
        }
        fprintf(lin, "%12.7f %12.7f %12.7f %12.7f %12.7f %12.7f", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
        fclose(lin);
        fclose(lin2);
    }
}


void MD_INFORMATION::Update_Volume(double factor)
{
    sys.box_length = factor * sys.box_length;
    pbc.uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * sys.box_length;
    MD_Information_Crd_To_Uint_Crd();
}

void MD_INFORMATION::Update_Box_Length(VECTOR factor)
{
    sys.box_length.x = factor.x * sys.box_length.x;
    sys.box_length.y = factor.y * sys.box_length.y;
    sys.box_length.z = factor.z * sys.box_length.z;
    pbc.uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * sys.box_length;
    MD_Information_Crd_To_Uint_Crd();
}

float MD_INFORMATION::system_information::Get_Density()
{
    density = total_mass * 1e24f / 6.023e23f / Get_Volume();
    return density;
}

double MD_INFORMATION::system_information::Get_Current_Time()
{
    current_time = start_time + (double)dt_in_ps * steps;
    return current_time;
}

float MD_INFORMATION::system_information::Get_Volume()
{
    volume = box_length.x * box_length.y * box_length.z;
    return volume;
}

void MD_INFORMATION::MD_Information_Crd_To_Uint_Crd()
{
    Crd_To_Uint_Crd << <(atom_numbers + 127) / 128, 128 >> >
        (this->atom_numbers, 1.0f / sys.box_length, crd, uint_crd);
}


void MD_INFORMATION::NVE_iteration::Leap_Frog()
{
    if (max_velocity <= 0)
    {
        MD_Iteration_Leap_Frog << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt);
    }
    else
    {
        MD_Iteration_Leap_Frog_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt, max_velocity);
    }
}

void MD_INFORMATION::MINIMIZATION_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (md_info->mode == MINIMIZATION)
    {
        controller->printf("    Start initializing minimization:\n");
        Cuda_Malloc_Safely((void**)&last_crd, md_info->atom_numbers * sizeof(VECTOR));
        max_move = 0.1f;
        if (controller[0].Command_Exist("minimization_max_move"))
        {
            controller->Check_Float("minimization", "max_move", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            max_move = atof(controller[0].Command("minimization_max_move"));
        }
        controller->printf("        minimization max move is %f A\n", max_move);

        momentum_keep = 0;
        if (controller[0].Command_Exist("minimization_momentum_keep"))
        {
            controller->Check_Float("minimization", "momentum_keep", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            momentum_keep = atof(controller[0].Command("minimization_momentum_keep"));
        }
        controller->printf("        minimization momentum keep is %f\n", momentum_keep);

        dynamic_dt = 1;
        if (controller[0].Command_Exist("minimization_dynamic_dt"))
        {
            controller->Check_Int("minimization", "dynamic_dt", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
            dynamic_dt = atoi(controller[0].Command("minimization_dynamic_dt"));
        }
        controller->printf("        minimization dynamic dt is %d\n", dynamic_dt);

        if (dynamic_dt)
        {
            dt_decreasing_rate = 0.01;
            if (controller[0].Command_Exist("minimization_dt_decreasing_rate"))
            {
                controller->Check_Float("minimization", "dt_decreasing_rate", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_decreasing_rate = atof(controller[0].Command("minimization_dt_decreasing_rate"));
            }
            controller->printf("        minimization dt decreasing rate is %f\n", dt_decreasing_rate);

            dt_increasing_rate = 1.01 + rand() / RAND_MAX;
            if (controller[0].Command_Exist("minimization_dt_increasing_rate"))
            {
                controller->Check_Float("minimization", "dt_increasing_rate", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_increasing_rate = atof(controller[0].Command("minimization_dt_increasing_rate"));
            }
            controller->printf("        minimization dt increasing rate is %f\n", dt_increasing_rate);
    
           dt_factor = 1e-4 + 1e-2 * rand() / RAND_MAX;
            if (controller[0].Command_Exist("minimization_dt_factor"))
            {
                controller->Check_Float("minimization", "dt_factor", "MD_INFORMATION::MINIMIZATION_iteration::Initial");
                dt_factor = atof(controller[0].Command("minimization_dt_factor"));
            }
            controller->printf("        minimization dt factor is %f\n", dt_factor);
            last_decrease_step = 0;
            Cuda_Malloc_Safely((void**)&d_max_force, sizeof(float));
        }
        controller->printf("    End initializing minimization\n\n");
    }
}

static __global__ void find_max_frc(int atom_numbers,  VECTOR *frc,  float *d_max_force)
{
    __shared__ float maxf[1024];
    float temp_f, temp_max = 0;
    VECTOR temp_frc;
    for (int i = threadIdx.x; i < atom_numbers; i += blockDim.x)
    {
        temp_frc = frc[i];
        temp_f = norm3df(temp_frc.x, temp_frc.y, temp_frc.z);
        if (temp_f > temp_max)
        {
            temp_max = temp_f;
        }
    }
    maxf[threadIdx.x] = temp_max;
    __syncthreads();
    int delta = 1024;
    for (int i = 0; i < 10; i += 1)
    {
        delta >>= 1;
        if (threadIdx.x < delta)
        {
            maxf[threadIdx.x] = fmaxf(maxf[threadIdx.x], maxf[threadIdx.x + delta]);
        }
    }
    d_max_force[0] = maxf[0];
}

void MD_INFORMATION::MINIMIZATION_iteration::Gradient_Descent()
{
    if (dynamic_dt)
    {
        if (md_info->sys.steps != 1)
        {
            if (last_decrease_step == 0)
            {
                if (last_potential > md_info->sys.h_potential)
                {
                    md_info->dt *= dt_increasing_rate;
                }
                else
                {
                    if (md_info->dt > 1e-8f)
                    {
                        md_info->dt *= dt_decreasing_rate;
                    }
                    else
                    {
                        last_decrease_step = md_info->sys.steps;
                    }
                }
            }
            else
            {
                find_max_frc<<<1, 1024>>>(md_info->atom_numbers, md_info->frc, d_max_force);
                hipMemcpy(&h_max_force, d_max_force, sizeof(float), hipMemcpyDeviceToHost);
                md_info->dt = sqrtf(dt_factor / h_max_force);
                if (md_info->sys.steps - last_decrease_step > 100)
                {
                    last_decrease_step = 0;
                }
            }
        }
        
        if (isnan(md_info->sys.h_potential) || (last_potential != 0 && (md_info->sys.h_potential - last_potential) / fabsf(last_potential) > 10))
        {
            md_info->sys.h_potential = last_potential;
            hipMemcpy(md_info->crd, last_crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
            hipMemset(md_info->vel, 0, sizeof(VECTOR) * md_info->atom_numbers);
            if (md_info->dt > 1e-8)
            {
                md_info->dt *= dt_decreasing_rate;
            }
            else
            {
                last_decrease_step = md_info->sys.steps;
            }
        }
        else
        {
            last_potential = md_info->sys.h_potential;
            hipMemcpy(last_crd, md_info->crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
        }
    }

    if (max_move <= 0)
    {
        MD_Iteration_Gradient_Descent << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep);
    }
    else
    {
        MD_Iteration_Gradient_Descent_With_Max_Move << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
            (md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep, max_move);
    }
}

void MD_INFORMATION::MINIMIZATION_iteration::Check_Nan()
{
    if (dynamic_dt)
    {
        float temperature = md_info->res.Get_Residue_Temperature();
        if (isnan(temperature) || (last_temperature > 0.1 && temperature / last_temperature > 10) || (last_temperature < 0.01 && temperature > 0.1))
        {
            md_info->sys.h_potential = last_potential;
            hipMemcpy(md_info->crd, last_crd, sizeof(VECTOR) * md_info->atom_numbers, hipMemcpyDeviceToDevice);
            hipMemset(md_info->vel, 0, sizeof(VECTOR) * md_info->atom_numbers);
            if (md_info->dt > 1e-8)
            {
                md_info->dt *= dt_decreasing_rate;
            }
            else
            {
                last_decrease_step = md_info->sys.steps;
            }
        }
        last_temperature = temperature;
    }
}

void MD_INFORMATION::RERUN_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    this->md_info = md_info;
    if (md_info->mode == RERUN)
    {
        controller->printf("    Start initializing rerun:\n");
        if (!Open_File_Safely(&traj_file, controller->Command(TRAJ_COMMAND), "rb"))
        {
            controller->Throw_SPONGE_Error(spongeErrorMissingCommand,
                "MD_INFORMATION::RERUN_information::Initial",
                "Reason:\n\tno trajectory information found (command 'crd' needed)");
        }
        else
        {
            controller->printf("        Open rerun coordinate trajectory\n");
        }
        if (!Open_File_Safely(&box_file, controller->Command(BOX_TRAJ_COMMAND), "r"))
        {
            box_file = NULL;
        }
        else
        {
            controller->printf("        Open rerun box trajectory\n");
        }
        start_frame = 0;
        if (controller->Command_Exist("rerun_start"))
        {
            controller->Check_Float("rerun_start", "MD_INFORMATION::RERUN_information::Initial");
            start_frame = atoi(controller->Command("rerun_start"));
        }
        strip_frame = 0;
        if (controller->Command_Exist("rerun_strip"))
        {
            controller->Check_Float("rerun_strip", "MD_INFORMATION::RERUN_information::Initial");
            strip_frame = atoi(controller->Command("rerun_strip"));
        }
        md_info->sys.step_limit = INT_MAX;
        controller->printf("    End initializing rerun\n\n");
    }
}

void MD_INFORMATION::RERUN_information::Iteration(int strip)
{
    int n;
    if (strip < 0)
    {
        strip = this->strip_frame;
    }
    for (int i = 0; i < strip; i++)
    {
        n = fread(this->md_info->coordinate, sizeof(VECTOR), this->md_info->atom_numbers, traj_file);
        md_info->sys.steps += 1;
    }
    n = fread(this->md_info->coordinate, sizeof(VECTOR), this->md_info->atom_numbers, traj_file);
    if (n != this->md_info->atom_numbers)
    {
        md_info->sys.step_limit = md_info->sys.steps;
    }
    hipMemcpy(this->md_info->crd, this->md_info->coordinate, sizeof(VECTOR)* this->md_info->atom_numbers, hipMemcpyHostToDevice);
    if (box_file != NULL)
    {
        int ret = fscanf(box_file, "%f %f %f %*f %*f %*f", &box_length_change_factor.x, &box_length_change_factor.y, &box_length_change_factor.z);
        box_length_change_factor = box_length_change_factor / md_info->sys.box_length;  
    }
#ifndef NO_GLOBAL_CONTROLLER
    else
    {
        extern CONTROLLER controller;
        controller.Throw_SPONGE_Error(spongeErrorMissingCommand, "MD_INFORMATION::RERUN_information::Iteration", "box traj file needed for rerun\n");        
    }
#endif
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_1()
{
    MD_Iteration_Speed_Verlet_1 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
        (md_info->atom_numbers, 0.5*md_info->dt, md_info->dt, md_info->acc, md_info->vel, md_info->crd, md_info->frc);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_2()
{
    if (max_velocity <= 0)
    {
        MD_Iteration_Speed_Verlet_2 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
            (md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc);
    }
    else
    {
        MD_Iteration_Speed_Verlet_2_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
            (md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc, max_velocity);
    }
}


float MD_INFORMATION::system_information::Get_Total_Atom_Ek(int is_download)
{
    MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);
    if (is_download)
    {
        hipMemcpy(&h_sum_of_atom_ek, d_sum_of_atom_ek, sizeof(float), hipMemcpyDeviceToHost);
        return h_sum_of_atom_ek;
    }
    else
    {
        return 0;
    }    
}

float MD_INFORMATION::system_information::Get_Atom_Temperature()
{
    h_temperature = Get_Total_Atom_Ek() * 2. / CONSTANT_kB / freedom;
    return h_temperature;
}

float MD_INFORMATION::residue_information::Get_Total_Residue_Ek(int is_download)
{
    MD_Residue_Ek << <ceilf((float)residue_numbers / 32.), 32 >> >
        (residue_numbers, d_res_start, d_res_end, res_ek_energy,
        md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(residue_numbers, res_ek_energy, sigma_of_res_ek);
    if (is_download)
    {
        hipMemcpy(h_sigma_of_res_ek, sigma_of_res_ek, sizeof(float), hipMemcpyDeviceToHost);
        return h_sigma_of_res_ek[0];
    }
    else
    {
        return 0;
    }
}

float MD_INFORMATION::residue_information::Get_Residue_Temperature()
{
    h_temperature = Get_Total_Residue_Ek() * 2. / CONSTANT_kB / residue_numbers / 3;
    return h_temperature;
}

void MD_INFORMATION::residue_information::Residue_Crd_Map(VECTOR *no_wrap_crd, float scaler)
{
    Get_Center_Of_Mass << <20, 32 >> >(residue_numbers, d_res_start, d_res_end, no_wrap_crd, md_info->d_mass, d_mass_inverse, d_center_of_mass);
    Map_Center_Of_Mass << <20, { 32, 4 } >> >(residue_numbers, d_res_start, d_res_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}


void MD_INFORMATION::MD_Reset_Atom_Energy_And_Virial_And_Force()
{
    need_potential = 0;
    hipMemset(d_atom_energy, 0, sizeof(float)* atom_numbers);
    hipMemset(sys.d_potential, 0, sizeof(float));

    need_pressure = 0;
    hipMemset(d_atom_virial, 0, sizeof(float)* atom_numbers);
    hipMemset(sys.d_virial, 0, sizeof(float));

    hipMemset(frc, 0, sizeof(VECTOR)* atom_numbers);
}


void MD_INFORMATION::Calculate_Pressure_And_Potential_If_Needed(int is_download)
{
    if (need_pressure > 0)
    {
        sys.Get_Pressure(is_download);
    }
    if (need_potential > 0)
    {
        sys.Get_Potential(is_download);
    }
}

float MD_INFORMATION::system_information::Get_Pressure(int is_download)
{
    //计算动能
    MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
    Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);

    //计算维里
    Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_virial, d_virial);

    //合并起来
    Calculate_Pressure_Cuda << <1, 1 >> >(1.0 / Get_Volume(), d_sum_of_atom_ek, d_virial, d_pressure);

    if (is_download)
    {
        hipMemcpy(&h_pressure, d_pressure, sizeof(float), hipMemcpyDeviceToHost);
        return h_pressure;
    }
    else
    {
        return 0;
    }
}

float MD_INFORMATION::system_information::Get_Potential(int is_download)
{

    Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_energy, d_potential);

    if (is_download)
    {
        hipMemcpy(&h_potential, d_potential, sizeof(float), hipMemcpyDeviceToHost);
        return h_potential;
    }
    else
    {
        return 0;
    }
}

void MD_INFORMATION::MD_Information_Frc_Device_To_Host()
{
    hipMemcpy(this->force, this->frc, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
}

void MD_INFORMATION::MD_Information_Frc_Host_To_Device()
{
    hipMemcpy(this->frc, this->force, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
}

void MD_INFORMATION::Crd_Vel_Device_To_Host(int forced)
{
    if (output.current_crd_synchronized_step != sys.steps || forced)
    {    
        output.current_crd_synchronized_step = sys.steps;
        hipMemcpy(this->coordinate, this->crd, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
        hipMemcpy(this->velocity, this->vel, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
    }
}

void MD_INFORMATION::Step_Print(CONTROLLER* controller)
{
    if (mode == RERUN)
    {
        controller->Step_Print("frame", this->sys.steps);
    }
    else
    {
        controller->Step_Print("step", this->sys.steps);
        controller->Step_Print("time", this->sys.Get_Current_Time());
        controller->Step_Print("temperature", this->res.Get_Residue_Temperature());
    }

}

void MD_INFORMATION::Clear()
{

}


void MD_INFORMATION::molecule_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
    controller->printf("    Start initializing molecule list:\n");
    this->md_info = md_info;
    //分子拓扑是一个无向图，邻接表进行描述，通过排除表形成
    int edge_numbers = 2 * md_info->nb.excluded_atom_numbers;
    int *visited = NULL; //每个原子是否拜访过
    int *first_edge = NULL; //每个原子的第一个边（链表的头）
    int *edges = NULL;  //每个边的序号
    int *edge_next = NULL; //每个原子的边（链表结构）
    int *molecule_belongings = NULL; //每个原子属于的分子编号
    Malloc_Safely((void**)&visited, sizeof(int)*md_info->atom_numbers);
    Malloc_Safely((void**)&visited, sizeof(int)*md_info->atom_numbers);
    Malloc_Safely((void**)&first_edge, sizeof(int)*md_info->atom_numbers);
    Malloc_Safely((void**)&edges, sizeof(int)*edge_numbers);
    Malloc_Safely((void**)&edge_next, sizeof(int)*edge_numbers);
    Malloc_Safely((void**)&molecule_belongings, sizeof(int)*md_info->atom_numbers);
    //初始化链表
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        visited[i] = 0;
        first_edge[i] = -1;
    }
    int atom_i, atom_j, edge_count = 0;
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        atom_i = i;
        for (int j = md_info->nb.h_excluded_list_start[i] + md_info->nb.h_excluded_numbers[i] - 1; j >= md_info->nb.h_excluded_list_start[i]; j--) //这里使用倒序是因为链表构建是用的头插法
        {
            atom_j = md_info->nb.h_excluded_list[j];
            edge_next[edge_count] = first_edge[atom_i];
            first_edge[atom_i] = edge_count;
            edges[edge_count] = atom_j;
            edge_count++;
            edge_next[edge_count] = first_edge[atom_j];
            first_edge[atom_j] = edge_count;
            edges[edge_count] = atom_i;
            edge_count++;
        }
    }

    std::deque<int> queue;
    int atom;
    molecule_numbers = 0;
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        if (!visited[i])
        {
            visited[i] = 1;
            queue.push_back(i);
            while (!queue.empty())
            {
                atom = queue[0];
                queue.pop_front();
                molecule_belongings[atom] = molecule_numbers;
                edge_count = first_edge[atom];

                while (edge_count != -1)
                {
                    atom = edges[edge_count];
                    if (!visited[atom])
                    {
                        queue.push_back(atom);
                        visited[atom] = 1;
                    }
                    edge_count = edge_next[edge_count];
                }
            }
            molecule_numbers += 1;
        }
    }
    printf("        molecule numbers is %d\n", molecule_numbers);
    Malloc_Safely((void**)&h_mass, sizeof(float)*molecule_numbers);
    Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*molecule_numbers);
    Malloc_Safely((void**)&h_atom_start, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_atom_end, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_residue_start, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_residue_end, sizeof(int)*molecule_numbers);
    Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*molecule_numbers);

    Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_atom_start, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_atom_end, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_residue_start, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_residue_end, sizeof(int)*molecule_numbers);
    Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*molecule_numbers);

    int molecule_j = 0;
    h_atom_start[0] = 0;
    //该判断基于一个分子的所有原子一定在列表里是连续的
    for (int i = 0; i < md_info->atom_numbers; i++)
    {
        if (molecule_belongings[i] != molecule_j)
        {
            if (molecule_belongings[i] < molecule_j)
            {
                char error_reason[CHAR_LENGTH_MAX];
                sprintf(error_reason, "Reason:\n\tthe indexes of atoms in the same one molecule should be continous, and atom #%d is not right\n", i);
                controller->Throw_SPONGE_Error(spongeErrorValueErrorCommand, "MD_INFORMATION::molecule_information::Initial", error_reason);
            }
            h_atom_end[molecule_j] = i;
            molecule_j += 1;
            if (molecule_j < molecule_numbers)
                h_atom_start[molecule_j] = i;
        }
    }
    h_atom_end[molecule_numbers - 1] = md_info->atom_numbers;
    
    molecule_j = 0;
    h_residue_start[0] = 0;
    //该判断基于一个分子的所有残基一定在列表里是连续的，且原子在残基里也是连续的
    for (int i = 0; i < md_info->res.residue_numbers; i++)
    {
        if (md_info->res.h_res_start[i] == h_atom_end[molecule_j])
        {
            h_residue_end[molecule_j] = i;
            molecule_j += 1;
            if (molecule_j < molecule_numbers)
                h_residue_start[molecule_j] = i;
        }
    }
    h_residue_end[molecule_numbers - 1] = md_info->res.residue_numbers;

    for (int i = 0; i < molecule_numbers; i++)
    {
        h_mass[i] = 0;
        for (molecule_j = h_atom_start[i]; molecule_j < h_atom_end[i]; molecule_j++)
        {
            h_mass[i] += md_info->h_mass[molecule_j];
        }
        h_mass_inverse[i] = 1.0f / h_mass[i];
    }

    hipMemcpy(d_mass, h_mass, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_atom_start, h_atom_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_atom_end, h_atom_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_residue_start, h_residue_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
    hipMemcpy(d_residue_end, h_residue_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);

    free(visited);
    free(first_edge);
    free(edges);
    free(edge_next);
    free(molecule_belongings);
    controller->printf("    End initializing molecule list\n\n");
}

void MD_INFORMATION::molecule_information::Molecule_Crd_Map(VECTOR *no_wrap_crd, float scaler)
{
    //为了有一个分子有很多残基，而其他分子都很小这种情况的并行，先求残基的质心
    Get_Origin << <64, 128 >> >(md_info->res.residue_numbers, md_info->res.d_res_start, md_info->res.d_res_end, no_wrap_crd, md_info->d_mass, md_info->res.d_mass_inverse, md_info->res.d_center_of_mass);
    //再用残基的质心求分子的质心
    Get_Origin << <32, 64 >> >(molecule_numbers, d_residue_start, d_residue_end, md_info->res.d_center_of_mass, md_info->res.d_mass, d_mass_inverse, d_center_of_mass);

    Map_Center_Of_Mass << <20, { 32, 4 } >> >(molecule_numbers, d_atom_start, d_atom_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}


void MD_INFORMATION::molecule_information::Molecule_Crd_Map(VECTOR *no_wrap_crd, VECTOR scaler)
{
    //为了有一个分子有很多残基，而其他分子都很小这种情况的并行，先求残基的质心
    Get_Origin << <64, 128 >> >(md_info->res.residue_numbers, md_info->res.d_res_start, md_info->res.d_res_end, no_wrap_crd, md_info->d_mass, md_info->res.d_mass_inverse, md_info->res.d_center_of_mass);
    //再用残基的质心求分子的质心
    Get_Origin << <32, 64 >> >(molecule_numbers, d_residue_start, d_residue_end, md_info->res.d_center_of_mass, md_info->res.d_mass, d_mass_inverse, d_center_of_mass);

    Map_Center_Of_Mass << <20, { 32, 4 } >> >(molecule_numbers, d_atom_start, d_atom_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}
