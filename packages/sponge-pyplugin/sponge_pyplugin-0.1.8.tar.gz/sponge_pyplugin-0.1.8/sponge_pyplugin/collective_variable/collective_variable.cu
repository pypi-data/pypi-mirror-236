﻿#include "collective_variable.cuh"
#define CV_ERROR_CHAR_LENGTH_MAX 600

CV_MAP_TYPE* CV_MAP = new CV_MAP_TYPE;
CV_INSTANCE_TYPE* CV_INSTANCE_MAP = new CV_INSTANCE_TYPE;

void COLLECTIVE_VARIABLE_CONTROLLER::Initial(CONTROLLER* controller, int* no_direct_interaction_virtual_atom_numbers)
{
    controller->printf("START INITIALIZING CV CONTROLLER:\n");
    strcpy(module_name, "cv_controller");
    this->controller = controller;
    mdinfo = controller->mdinfo;
    if (controller->Command_Exist("cv_in_file"))
    {
		int CV_numbers = 0;
        Commands_From_In_File(controller);
		int cv_vatom_count = 0;
		for (StringMap::iterator iter = this->commands.begin(); iter != this->commands.end(); iter++)
		{
			int i = iter->first.rfind("vatom_type");
			if (i > 0 && i == (iter->first.length() - 10))
			{
				cv_vatom_name[iter->first.substr(0, i - 1)] = cv_vatom_count;
				cv_vatom_count += 1;
				no_direct_interaction_virtual_atom_numbers[0]++;
			}
			i = iter->first.rfind("CV_type");
			if (i > 0 && i == (iter->first.length() - 7))
			{
				CV_numbers++;
			}
		}
		printf("    %d CV defined\n", CV_numbers);
		printf("    %d cv virtual atoms\n", no_direct_interaction_virtual_atom_numbers[0]);
        is_initialized = 1;
		controller->printf("END INITIALIZING CV CONTROLLER\n\n");
    }
	else
	{
		controller->printf("CV CONTROLLER IS NOT INITIALIZING\n\n");
	}
    
}

static int read_one_line(FILE* In_File, char* line, char* ender)
{
    int line_count = 0;
    int ender_count = 0;
    char c;
    while ((c = getc(In_File)) != EOF)
    {
        if (line_count == 0 && (c == '\t' || c == ' '))
        {
            continue;
        }
        else if (c != '\n' && c != ',' && c != '{' && c != '}' && c != '\r')
        {
            line[line_count] = c;
            line_count += 1;
        }
        else
        {
            ender[ender_count] = c;
            ender_count += 1;
            break;
        }
    }
    while ((c = getc(In_File)) != EOF)
    {
        if (c == ' ' || c == '\t')
        {
            continue;
        }
        else if (c != '\n' && c != ',' && c != '{' && c != '}' && c != '\r')
        {
            fseek(In_File, -1, SEEK_CUR);
            break;
        }
        else
        {
            ender[ender_count] = c;
            ender_count += 1;
        }
    }
    line[line_count] = 0;
    ender[ender_count] = 0;
    if (line_count == 0 && ender_count == 0)
    {
        return EOF;
    }
    return 1;
}

void COLLECTIVE_VARIABLE_CONTROLLER::Commands_From_In_File(CONTROLLER* controller)
{
    FILE* In_File = NULL;
    if (controller->Command_Exist("cv_in_file"))
    {
        Open_File_Safely(&In_File, controller->Command("cv_in_file"), "r");
    } 
    if (In_File != NULL)
    {
        char line[CHAR_LENGTH_MAX];
        char prefix[CHAR_LENGTH_MAX] = { 0 };
        char ender[CHAR_LENGTH_MAX];
        while (true)
        {
            if (read_one_line(In_File, line, ender) == EOF)
            {
                break;
            }
            if (line[0] == '#')
            {
                if (line[1] == '#')
                {
                    if (strchr(ender, '{') != NULL)
                    {
                        int scanf_ret = sscanf(line, "%s", prefix);
                    }
                    if (strchr(ender, '}') != NULL)
                    {
                        prefix[0] = 0;
                    }
                }
                if (strchr(ender, '\n') == NULL)
                {
                    int scanf_ret = fscanf(In_File, "%*[^\n]%*[\n]");
                    fseek(In_File, -1, SEEK_CUR);
                }
            }
            else if (strchr(ender, '{') != NULL)
            {
                int scanf_ret = sscanf(line, "%s", prefix);
            }
            else
            {
                Get_Command(line, prefix);
                line[0] = 0;
            }
            if (strchr(ender, '}') != NULL)
            {
                prefix[0] = 0;
            }
        }
    }
}

void COLLECTIVE_VARIABLE_CONTROLLER::Input_Check()
{
	for (int i = 0; i < print_cv_list.size(); i++)
	{
		controller->Step_Print_Initial(print_cv_list[i]->module_name, "%.4f");
	}
    if (!(Command_Exist("dont_check_input") && atoi(Command("dont_check_input"))))
    {
        int no_warning = 0;
        for (CheckMap::iterator iter = command_check.begin(); iter != command_check.end(); iter++)
        {
            if (iter->second == 1)
            {
                printf("Warning: CV command '%s' is set, but never used.\n", iter->first.c_str());
                no_warning += 1;
            }
        }
        for (CheckMap::iterator iter = choice_check.begin(); iter != choice_check.end(); iter++)
        {
            if (iter->second == 2)
            {
                printf("Warning: the value '%s' of CV command '%s' matches none of the choices.\n", this->commands[iter->first].c_str(), iter->first.c_str());
                no_warning += 1;
            }
            else if (iter->second == 3)
            {
                printf("Warning: CV command '%s' is not set.\n", iter->first.c_str());
                no_warning += 1;
            }
        }
        if (no_warning)
        {
            printf("\nWarning: CV inputs raised %d warning(s). If You know WHAT YOU ARE DOING, press any key to continue. Set dont_check_input = 1 to disable this warning.\n", no_warning);
            getchar();
        }
    }
}

void COLLECTIVE_VARIABLE_CONTROLLER::Print_Initial()
{
    if (!is_initialized)
        return;
    controller->printf("START INITIALIZING CV PRINTER:\n");
	print_cv_list = Ask_For_CV("print", 0);
	for (int i = 0; i < print_cv_list.size(); i++)
	{
		if (controller->outputs_content.count(print_cv_list[i]->module_name))
		{
			std::string error_reason = "Reason:\n\tthe name of the CV '";
			error_reason += print_cv_list[i]->module_name;
			error_reason += "' to print is the same with a built-in output\n";
			controller->Throw_SPONGE_Error(spongeErrorConflictingCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Print_Initial", error_reason.c_str());
		}
	}
    controller->printf("END INITIALIZING CV PRINTER\n\n");
}

void COLLECTIVE_VARIABLE_CONTROLLER::Step_Print(int step, int atom_numbers, UNSIGNED_INT_VECTOR* uint_crd, VECTOR scaler, VECTOR* crd, VECTOR box_length)
{
	for (int i = 0; i < print_cv_list.size(); i++)
	{
		print_cv_list[i]->Compute(atom_numbers, uint_crd, scaler, crd, box_length, CV_NEED_CPU_VALUE, step + 1);
	}
	for (int i = 0; i < print_cv_list.size(); i++)
	{
		hipStreamSynchronize(print_cv_list[i]->hip_stream);
		controller->Step_Print(print_cv_list[i]->module_name, print_cv_list[i]->value); 
	}
	
}

COLLECTIVE_VARIABLE_PROTOTYPE* COLLECTIVE_VARIABLE_CONTROLLER::get_CV(const char* cv_name)
{
    if (!is_initialized)
    {
		this->Throw_SPONGE_Error(spongeErrorMissingCommand, "COLLECTIVE_VARIABLE_CONTROLLER::get_CV", "Reason:\n\tcommand 'cv_in_file' is not set\n");
    }
    if (CV_INSTANCE_MAP->count(cv_name))
    {
        return CV_INSTANCE_MAP[0][cv_name];
    }
    if (Command_Exist(cv_name, "CV_type"))
    {
        char cv_type[CHAR_LENGTH_MAX];
        strcpy(cv_type, Command(cv_name, "CV_type"));
        if (CV_MAP->count(cv_type))
        {
            COLLECTIVE_VARIABLE_PROTOTYPE *cv = CV_MAP[0][cv_type](this, cv_name);
            CV_INSTANCE_MAP[0][cv_name] = cv;
            return CV_INSTANCE_MAP[0][cv_name];
        }
        else
        {
			char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
			sprintf(error_reason, "Reason:\n\tthe type '%s' of the CV '%s' is undefined\n", cv_type, cv_name);
			this->Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::get_CV", error_reason);
        }
        
    }
    else
    {
		char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
		sprintf(error_reason, "Reason:\n\tthe type of the CV '%s' is undefined\n", cv_name);
		this->Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::get_CV", error_reason);
    }
    return 0;
}

CV_LIST COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_CV(const char* name, int N, float verbose_level, int layout)
{
    CV_LIST cv_list;
    int CV_numbers = 0;
    char command[CHAR_LENGTH_MAX];
	sprintf(command, "%s_CV", name);
	if (Command_Exist(command))
	{
		strcpy(command, Original_Command(command));
		char *cv_name = strtok(command, " ");
		while (cv_name != NULL)
		{
			CV_numbers += 1;
			cv_name = strtok(NULL, " ");
		}
	}
	if (N > 0 && CV_numbers != N)
	{
		char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
		sprintf(error_reason, "Reason:\n\t%d CV(s) should be given to %s, but %d found\n", N, name, CV_numbers);
		Throw_SPONGE_Error(spongeErrorValueErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_CV", error_reason);
	}
	else if (N <= 0 && CV_numbers < -N)
	{
		char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
		sprintf(error_reason, "Reason:\n\tat least %d CV(s) should be given to %s, but only %d found\n", N, name, CV_numbers);
		Throw_SPONGE_Error(spongeErrorValueErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_CV", error_reason);
	}
	if (verbose_level > -2)
	{
		for (int i = 0; i < layout; i++)
		{
			printf("    ");
		}
		printf("%d CV(s) found for %s\n", CV_numbers, name);
	}
	sprintf(command, "%s_CV", name);
	//c的strtok在此处线程不安全，只能用C++的函数分割字符串
	COLLECTIVE_VARIABLE_PROTOTYPE *cv;
	std::string value = Original_Command(command);
	auto start = value.find_first_not_of(' ', 0);
	auto stop = value.find_first_of(' ', start);
    for (int i = 0; i < CV_numbers; i++)
    {
		strcpy(command, value.substr(start, stop - start).c_str());
		if (verbose_level > -1)
		{
			for (int ii = 0; ii < layout; ii++)
			{
				printf("    ");
			}
			printf("    CV %d: %s\n", i, command);
		}
			
		cv = get_CV(command);
		if (verbose_level > -1)
		{
			for (int ii = 0; ii < layout; ii++)
			{
				printf("    ");
			}
			printf("        type of '%s' is '%s'\n", command, cv->type_name);
		}
		cv_list.push_back(cv);
		start = value.find_first_not_of(" ", stop);
		stop = value.find_first_of(" ", start);
    }
    return cv_list;
}

int* COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Int_Parameter(const char* name, const char* parameter_name, int N, int layout,
	bool raise_error_when_missing, int default_value, float verbose_level, const char* unit)
{
	if (unit == NULL)
	{
		unit = "";
	}
	int* t;
	char command[CHAR_LENGTH_MAX];
	sprintf(command, "%s_%s", name, parameter_name);
	if (!this->Command_Exist(command))
	{
		if (raise_error_when_missing)
		{
			char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
			sprintf(error_reason, "Reason:\n\tno parameter %s found for %s\n", parameter_name, name);
			Throw_SPONGE_Error(spongeErrorMissingCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Int_Parameter", error_reason);
		}
		else
		{
			strcpy(command, "");
		}
	}
	else
	{
		strcpy(command, this->Original_Command(command));
	}
	if (verbose_level > -2)
	{
		for (int _lay = 0; _lay < layout; _lay++)
			this->printf("    ");
		this->printf("reading %d %s(s) for %s\n", N, parameter_name, name);
	}
	Malloc_Safely((void**)&t, sizeof(int)* N);
	char *token = strtok(command, " ");
	for (int i = 0; i < N; i++)
	{
		if (token == NULL)
		{
			if (raise_error_when_missing)
			{
				char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
				sprintf(error_reason, "Reason:\n\tthe number of parameter should be %d, but %d found\n", N, i);
				this->Throw_SPONGE_Error(spongeErrorValueErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Int_Parameter", error_reason);
			}
			else
			{
				t[i] = default_value;
				if (verbose_level > -1)
				{
					for (int _lay = 0; _lay < layout; _lay++)
						this->printf("    ");
					this->printf("    %s %d: %d %s (from default value)\n", parameter_name, i, t[i], unit);
				}

			}
		}
		else
		{
			if (cv_vatom_name.count(token))
			{
				t[i] = cv_vatom_name[token] + atom_numbers;
			}
			else if (controller->Command_Exist(token))
			{
				controller->Check_Int(token, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Int_Parameter");
				t[i] = atoi(controller->Command(token));
			}
			else
			{
				if (!is_str_int(token))
				{
					char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
					sprintf(error_reason, "Reason:\n\t the %d-th value '%s' of the command '%s' is not an int\n", i, token, command);
					controller->Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Int_Parameter", error_reason);
				}
				t[i] = atoi(token);
			}
			if (verbose_level > -1)
			{
				for (int _lay = 0; _lay < layout; _lay++)
					this->printf("    ");
				this->printf("    %s %d: %d %s\n", parameter_name, i, t[i], unit);
			}
		}
		token = strtok(NULL, " ");
	}
	return t;
}

float* COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Float_Parameter(const char* name, const char* parameter_name, int N, int layout,
    bool raise_error_when_missing, float default_value, float verbose_level, const char* unit)
{
	if (unit == NULL)
	{
		unit = "";
	}
	float* t;
	char command[CHAR_LENGTH_MAX];
	sprintf(command, "%s_%s", name, parameter_name);
	if (!this->Command_Exist(command))
	{
		if (raise_error_when_missing)
		{
			char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
			sprintf(error_reason, "Reason:\n\tno parameter %s found for %s\n", parameter_name, name);
			Throw_SPONGE_Error(spongeErrorMissingCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Float_Parameter", error_reason);
		}
		else
		{
			strcpy(command, "");
		}
	}
	else
	{
		strcpy(command, this->Original_Command(command));
	}
	if (verbose_level > -2)
	{
		for (int _lay = 0; _lay < layout; _lay++)
			this->printf("    ");
		this->printf("reading %d %s(s) for %s\n", N, parameter_name, name);
	}
	Malloc_Safely((void**)&t, sizeof(float)* N);
	char *token = strtok(command, " ");
	for (int i = 0; i < N; i++)
	{
		if (token == NULL)
		{
			if (raise_error_when_missing)
			{
				char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
				sprintf(error_reason, "Reason:\n\tthe number of parameter should be %d, but %d found\n", N, i);
				Throw_SPONGE_Error(spongeErrorValueErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Float_Parameter", error_reason);
			}
			else
			{
				t[i] = default_value;
				if (verbose_level > -1)
				{
					for (int _lay = 0; _lay < layout; _lay++)
						this->printf("    ");
					this->printf("    %s %d: %d %s (from default value)\n", parameter_name, i, t[i], unit);
				}

			}
		}
		else
		{
			if (controller->Command_Exist(token))
			{
				controller->Check_Float(token, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Float_Parameter");
				t[i] = atof(controller->Command(token));
			}
			else
			{
				if (!is_str_float(token))
				{
					char error_reason[CV_ERROR_CHAR_LENGTH_MAX];
					sprintf(error_reason, "Reason:\n\t the %d-th value '%s' of the command '%s' is not a float\n", i, token, command);
					controller->Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Float_Parameter", error_reason);
				}
				t[i] = atof(token);
			}
			if (verbose_level > -1)
			{
				for (int _lay = 0; _lay < layout; _lay++)
					this->printf("    ");
				this->printf("    %s %d: %f %s\n", parameter_name, i, t[i], unit);
			}
		}
		token = strtok(NULL, " ");
	}
	return t;
}

std::vector<std::string> COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_String_Parameter(const char* name, const char* parameter_name, int N, int layout,
	bool raise_error_when_missing, const char* default_value, float verbose_level, const char* unit)
{
	if (unit == NULL)
	{
		unit = "";
	}
	std::vector<std::string> t;
	char command[CHAR_LENGTH_MAX];
	sprintf(command, "%s_%s", name, parameter_name);
	if (!this->Command_Exist(command))
	{
		if (raise_error_when_missing)
		{
			char error_reason[CHAR_LENGTH_MAX];
			sprintf(error_reason, "Reason:\n\tno parameter %s found for %s\n", parameter_name, name);
			Throw_SPONGE_Error(spongeErrorMissingCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_String_Parameter", error_reason);
		}
		else
		{
			strcpy(command, "");
		}
	}
	else
	{
		strcpy(command, this->Original_Command(command));
	}
	if (verbose_level > -2)
	{
		for (int _lay = 0; _lay < layout; _lay++)
			this->printf("    ");
		this->printf("reading %d %s(s) for %s\n", N, parameter_name, name);
	}
	char* token = strtok(command, " ");
	for (int i = 0; i < N; i++)
	{
		if (token == NULL)
		{
			if (raise_error_when_missing)
			{
				char error_reason[CHAR_LENGTH_MAX];
				sprintf(error_reason, "Reason:\n\tthe number of parameter should be %d, but %d found\n", N, i);
				Throw_SPONGE_Error(spongeErrorValueErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_String_Parameter", error_reason);
			}
			else
			{
				t.push_back(std::string(default_value));
				if (verbose_level > -1)
				{
					for (int _lay = 0; _lay < layout; _lay++)
						this->printf("    ");
					this->printf("    %s %d: %s %s (from default value)\n", parameter_name, i, t[i].c_str(), unit);
				}

			}
		}
		else
		{
			if (controller->Command_Exist(token))
			{
				t.push_back(std::string(controller->Command(token)));
			}
			else
			{
				t.push_back(std::string(token));
			}
			if (verbose_level > -1)
			{
				for (int _lay = 0; _lay < layout; _lay++)
					this->printf("    ");
				this->printf("    %s %d: %s %s\n", parameter_name, i, t[i].c_str(), unit);
			}
		}
		token = strtok(NULL, " ");
	}
	return t;
}

std::vector<int> COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter(const char* name, const char* parameter_name)
{
	std::vector<int> ints;
	char out[CHAR_LENGTH_MAX];
	std::string file_name = parameter_name;
	file_name += "_in_file";
	if (Command_Exist(name, parameter_name))
	{
		std::string strs = Original_Command(name, parameter_name);
		std::istringstream ss(strs);
		while (ss >> out)
		{
			if (cv_vatom_name.count(out))
			{
				ints.push_back(cv_vatom_name[out] + atom_numbers);
			}
			else if (controller->Command_Exist(out))
			{
				controller->Check_Int(out, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter");
				ints.push_back(atoi(controller->Command(out)));
			}
			else
			{
				if (!is_str_int(out))
				{
					Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter",
						"Reason:\n\tone of the value is not an int\n");
				}
				ints.push_back(atoi(out));
			}
		}
	}
	else if (Command_Exist(name, file_name.c_str()))
	{
		std::ifstream ss(Command(name, file_name.c_str()));
		while (ss >> out)
		{
			if (cv_vatom_name.count(out))
			{
				ints.push_back(cv_vatom_name[out] + atom_numbers);
			}
			else if (controller->Command_Exist(out))
			{
				controller->Check_Int(out, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter");
				ints.push_back(atoi(controller->Command(out)));
			}
			else
			{
				if (!is_str_int(out))
				{
					Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter",
						"Reason:\n\tone of the value is not an int\n");
				}
				ints.push_back(atoi(out));
			}
		}
	}
	return ints;
}

std::vector<float> COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Float_Parameter(const char* name, const char* parameter_name)
{
	std::vector<float> floats;
	char out[CHAR_LENGTH_MAX];
	std::string file_name = parameter_name;
	file_name += "_in_file";
	std::string error_reason = "Reason:\n\tone value of the parameter '";
	error_reason += parameter_name;
	error_reason += "' is not a float";
	if (Command_Exist(name, parameter_name))
	{
		std::string strs = Original_Command(name, parameter_name);
		std::istringstream ss(strs);
		while (ss >> out)
		{
			if (controller->Command_Exist(out))
			{
				controller->Check_Float(out, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter");
				floats.push_back(atof(controller->Command(out)));
			}
			else
			{
				if (!is_str_float(out))
				{
					Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter", error_reason.c_str());
				}
				floats.push_back(atof(out));
			}
		}
	}
	else if (Command_Exist(name, file_name.c_str()))
	{
		std::ifstream ss(Command(name, file_name.c_str()));
		while (ss >> out)
		{
			if (controller->Command_Exist(out))
			{
				controller->Check_Float(out, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter");
				floats.push_back(atof(controller->Command(out)));
			}
			else
			{
				if (!is_str_float(out))
				{
					Throw_SPONGE_Error(spongeErrorTypeErrorCommand, "COLLECTIVE_VARIABLE_CONTROLLER::Ask_For_Indefinite_Length_Int_Parameter", error_reason.c_str());
				}
				floats.push_back(atof(out));
			}
		}
	}
	return floats;
}

int COLLECTIVE_VARIABLE_PROTOTYPE::Check_Whether_Computed_At_This_Step(int step, int need)
{
    if ((need & CV_NEED_CPU_VALUE) && (last_update_step[CV_NEED_CPU_VALUE] == step))
        need &= ~CV_NEED_CPU_VALUE;
    if ((need & CV_NEED_GPU_VALUE) && (last_update_step[CV_NEED_GPU_VALUE] == step))
        need &= ~CV_NEED_GPU_VALUE;
    if ((need & CV_NEED_CRD_GRADS) && (last_update_step[CV_NEED_CRD_GRADS] == step))
        need &= ~CV_NEED_CRD_GRADS;
    if ((need & CV_NEED_BOX_GRADS) && (last_update_step[CV_NEED_BOX_GRADS] == step))
        need &= ~CV_NEED_BOX_GRADS;
    return need;
}

void COLLECTIVE_VARIABLE_PROTOTYPE::Record_Update_Step_Of_Slow_Computing_CV(int step, int need)
{
    if (need & CV_NEED_CPU_VALUE)
        last_update_step[CV_NEED_CPU_VALUE] = step;
    if (need & CV_NEED_CRD_GRADS)
        last_update_step[CV_NEED_CRD_GRADS] = step;
    if (need & CV_NEED_GPU_VALUE)
        last_update_step[CV_NEED_GPU_VALUE] = step;
    if (need & CV_NEED_BOX_GRADS)
        last_update_step[CV_NEED_BOX_GRADS] = step;
}

void COLLECTIVE_VARIABLE_PROTOTYPE::Record_Update_Step_Of_Fast_Computing_CV(int step, int need)
{
	last_update_step[CV_NEED_CRD_GRADS] = step;
	last_update_step[CV_NEED_GPU_VALUE] = step;
	last_update_step[CV_NEED_BOX_GRADS] = step;
	last_update_step[CV_NEED_CPU_VALUE] = step;
}

void COLLECTIVE_VARIABLE_PROTOTYPE::Super_Initial(COLLECTIVE_VARIABLE_CONTROLLER* manager, int atom_numbers, const char* module_name)
{
    strcpy(this->module_name, module_name);
    Cuda_Malloc_Safely((void**)&crd_grads, sizeof(VECTOR) * atom_numbers);
	Cuda_Malloc_Safely((void**)&box_grads, sizeof(VECTOR));
    Cuda_Malloc_Safely((void**)&d_value, sizeof(float));
	hipMemset(crd_grads, 0, sizeof(VECTOR)* atom_numbers);
	hipMemset(box_grads, 0, sizeof(VECTOR));
	hipStreamCreate(&hip_stream);
    last_update_step[CV_NEED_GPU_VALUE] = -1;
    last_update_step[CV_NEED_CPU_VALUE] = -1;
    last_update_step[CV_NEED_CRD_GRADS] = -1;
    last_update_step[CV_NEED_BOX_GRADS] = -1;
}

void COLLECTIVE_VARIABLE_PROTOTYPE::Initial(COLLECTIVE_VARIABLE_CONTROLLER* manager, int atom_numbers, const char* module_name)
{
	Super_Initial(manager, atom_numbers, module_name);
}
