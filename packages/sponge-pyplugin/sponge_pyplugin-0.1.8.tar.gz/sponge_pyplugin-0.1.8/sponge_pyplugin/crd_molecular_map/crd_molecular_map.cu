#include "hip/hip_runtime.h"
﻿#include "crd_molecular_map.cuh"

static __global__ void Calculate_No_Wrap_Crd_CUDA
(const int atom_numbers, const INT_VECTOR *box_map_times, const VECTOR box, const VECTOR *crd,
VECTOR *nowrap_crd)
{
    for (int i = threadIdx.x; i < atom_numbers; i = i + blockDim.x)
    {
        nowrap_crd[i].x = (float)box_map_times[i].int_x*box.x + crd[i].x;
        nowrap_crd[i].y = (float)box_map_times[i].int_y*box.y + crd[i].y;
        nowrap_crd[i].z = (float)box_map_times[i].int_z*box.z + crd[i].z;
    }
}

static __global__ void Refresh_BoxMapTimes_CUDA
(const int atom_numbers, const VECTOR box_length_inverse, const VECTOR *crd,
INT_VECTOR *box_map_times, VECTOR *old_crd)
{
    VECTOR crd_i, old_crd_i;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < atom_numbers; i += gridDim.x * blockDim.x)
    {
        crd_i = crd[i];
        old_crd_i = old_crd[i];
        box_map_times[i].int_x += floor((old_crd_i.x - crd_i.x) * box_length_inverse.x + 0.5);
        box_map_times[i].int_y += floor((old_crd_i.y - crd_i.y) * box_length_inverse.y + 0.5);
        box_map_times[i].int_z += floor((old_crd_i.z - crd_i.z) * box_length_inverse.z + 0.5);
        old_crd[i] = crd_i;
    }
}

static void Move_Crd_Nearest_From_Connectivity(CPP_ATOM_GROUP mol_atoms, VECTOR *crd, INT_VECTOR *box_map_times, const VECTOR box_length, std::vector<int> periodic_molecules)
{
    std::vector<int> atoms;
    for (int i = 0; i < mol_atoms.size(); i++)
    {
        if (periodic_molecules[i])
        {
            continue;
        }
        atoms = mol_atoms[i];
        for (int j = 1; j < atoms.size(); j++)
        {
            int atom = atoms[j];
            int atom_front = atoms[j - 1];
            int map_x = floorf((crd[atom_front].x - crd[atom].x) / box_length.x + 0.5);
            crd[atom].x = crd[atom].x + map_x * box_length.x;
            map_x = floorf((crd[atom_front].y - crd[atom].y) / box_length.y + 0.5);
            crd[atom].y = crd[atom].y + map_x * box_length.y;
            map_x = floorf((crd[atom_front].z - crd[atom].z) / box_length.z + 0.5);
            crd[atom].z = crd[atom].z + map_x * box_length.z;
        }
    }
}

static void Get_Molecule_Atoms(CONTROLLER* controller, int atom_numbers, CONECT connectivity, CPP_ATOM_GROUP &mol_atoms)
{
    //分子拓扑是一个无向图，邻接表进行描述，通过排除表形成
    int edge_numbers = 0;
    for (int i = 0; i < atom_numbers; i++)
    {
        edge_numbers += connectivity[i].size();
    }
    int* visited = NULL; //每个原子是否拜访过
    int* first_edge = NULL; //每个原子的第一个边（链表的头）
    int* edges = NULL;  //每个边的序号
    int* edge_next = NULL; //每个原子的边（链表结构）
    Malloc_Safely((void**)&visited, sizeof(int) * atom_numbers);
    Malloc_Safely((void**)&first_edge, sizeof(int) * atom_numbers);
    Malloc_Safely((void**)&edges, sizeof(int) * edge_numbers);
    Malloc_Safely((void**)&edge_next, sizeof(int) * edge_numbers);
    //初始化链表
    for (int i = 0; i < atom_numbers; i++)
    {
        visited[i] = 0;
        first_edge[i] = -1;
    }
    int atom_i, atom_j, edge_count = 0;
    for (int atom_i = 0; atom_i < atom_numbers; atom_i++)
    {
        std::set<int> conect_i = connectivity[atom_i];
        for (auto iter = conect_i.begin(); iter != conect_i.end(); iter++)
        {
            atom_j = *iter;
            edge_next[edge_count] = first_edge[atom_i];
            first_edge[atom_i] = edge_count;
            edges[edge_count] = atom_j;
            edge_count++;
        }
    }
    if (controller->Command_Exist("make_output_whole"))
    {
        char temp[CHAR_LENGTH_MAX];
        strcpy(temp, controller->Original_Command("make_output_whole"));
        char* word = strtok(temp, " -");
        while (word != NULL)
        {
            atom_i = atoi(word);
            word = strtok(NULL, "-");
            if (word == NULL)
            {
                controller->Throw_SPONGE_Error(spongeErrorValueErrorCommand, "Move_Crd_Nearest_From_Exclusions_Host",
                    "Reason:\n\t'make_output_whole' should provide atoms in the format of atom_i-atom_j");
            }
            atom_j = atoi(word);
            edge_next[edge_count] = first_edge[atom_i];
            first_edge[atom_i] = edge_count;
            edges[edge_count] = atom_j;
            edge_count++;
            edge_next[edge_count] = first_edge[atom_j];
            first_edge[atom_j] = edge_count;
            edges[edge_count] = atom_i;
            edge_count++;
            word = strtok(NULL, " ");
        }
    }
    std::deque<int> queue;
    int atom;
    for (int i = 0; i < atom_numbers; i++)
    {
        if (!visited[i])
        {
            std::vector<int> atoms;
            visited[i] = 1;
            queue.push_back(i);
            while (!queue.empty())
            {
                atom = queue[0];
                atoms.push_back(atom);
                queue.pop_front();
                edge_count = first_edge[atom];
                while (edge_count != -1)
                {
                    atom = edges[edge_count];
                    if (!visited[atom])
                    {
                        queue.push_back(atom);
                        visited[atom] = 1;
                    }
                    edge_count = edge_next[edge_count];
                }
            }
            mol_atoms.push_back(atoms);
        }
    }
    free(visited);
    free(first_edge);
    free(edges);
    free(edge_next);
}

void CoordinateMolecularMap::Record_Box_Map_Times_Host(int atom_numbers, VECTOR *crd, VECTOR *old_crd, INT_VECTOR *box_map_times, VECTOR box)
{
    for (int i = 0; i < atom_numbers; i = i + 1)
    {
        box_map_times[i].int_x += floor((old_crd[i].x - crd[i].x) / box_length.x + 0.5);
        box_map_times[i].int_y += floor((old_crd[i].y - crd[i].y) / box_length.y + 0.5);
        box_map_times[i].int_z += floor((old_crd[i].z - crd[i].z) / box_length.z + 0.5);
    }
}

std::vector<int> Check_Periodic_Molecules(CPP_ATOM_GROUP mol_atoms, const VECTOR* crd, const VECTOR box_length)
{
    std::vector<int> periodic_mols;
    std::vector<int> atoms;
    VECTOR abosolute_dr;
    for (int i = 0; i < mol_atoms.size(); i++)
    {
        atoms = mol_atoms[i];
        abosolute_dr.x = 0;
        abosolute_dr.y = 0;
        abosolute_dr.z = 0;
        for (int j = 1; j < atoms.size(); j++)
        {
            int atom = atoms[j];
            int atom_front = atoms[j-1];
            abosolute_dr = abosolute_dr + Get_Periodic_Displacement(crd[atom], crd[atom_front], box_length);
        }
        periodic_mols.push_back(abosolute_dr.x * abosolute_dr.x >= box_length.x * box_length.x
            || abosolute_dr.y * abosolute_dr.y >= box_length.y * box_length.y
            || abosolute_dr.z * abosolute_dr.z >= box_length.z * box_length.z); 
    }
    return periodic_mols;
}

void CoordinateMolecularMap::Initial(CONTROLLER *controller, int atom_numbers, VECTOR box_length, VECTOR *crd, 
    CONECT connectivity, const char *module_name)
{
    if (module_name == NULL)
    {
        strcpy(this->module_name, "crd_mole_wrap");
    }
    else
    {
        strcpy(this->module_name, module_name);
    }

    this->atom_numbers = atom_numbers;
    this->box_length = box_length;
    
    Cuda_Malloc_Safely((void**)&nowrap_crd, sizeof(VECTOR)*atom_numbers);
    Cuda_Malloc_Safely((void**)&old_crd, sizeof(VECTOR)*atom_numbers);
    Cuda_Malloc_Safely((void**)&box_map_times, sizeof(INT_VECTOR)*atom_numbers);

    Malloc_Safely((void**)&h_nowrap_crd, sizeof(VECTOR)*atom_numbers);
    Malloc_Safely((void**)&h_old_crd, sizeof(VECTOR)*atom_numbers);
    Malloc_Safely((void**)&h_box_map_times, sizeof(INT_VECTOR)*atom_numbers);
    hipMemcpy(h_nowrap_crd, crd, sizeof(VECTOR) * atom_numbers, hipMemcpyDeviceToHost);
    for (int i = 0; i < atom_numbers; i = i + 1)
    {
        h_old_crd[i] = h_nowrap_crd[i];
        h_box_map_times[i].int_x = 0;
        h_box_map_times[i].int_y = 0;
        h_box_map_times[i].int_z = 0;
    }
    if (controller[0].Command_Exist("molecule_map_output"))
    {
        controller->Warn("'molecule_map_output' is a deprecated command since version 1.4");
    }
    Get_Molecule_Atoms(controller, atom_numbers, connectivity, molecule_atoms);
    periodic_molecules = Check_Periodic_Molecules(molecule_atoms, h_nowrap_crd, box_length);
    Move_Crd_Nearest_From_Connectivity(molecule_atoms, h_nowrap_crd, h_box_map_times, box_length, periodic_molecules);
    //使用cuda内部函数，给出占用率最大的block和thread参数
    hipOccupancyMaxPotentialBlockSize(&blocks_per_grid, &threads_per_block, Refresh_BoxMapTimes_CUDA, 0, 0);

    hipMemcpy(nowrap_crd, h_nowrap_crd, sizeof(VECTOR)*atom_numbers, hipMemcpyHostToDevice);
    hipMemcpy(old_crd, h_old_crd, sizeof(VECTOR)*atom_numbers, hipMemcpyHostToDevice);
    hipMemcpy(box_map_times, h_box_map_times, sizeof(INT_VECTOR)*atom_numbers, hipMemcpyHostToDevice);
    is_initialized = 1;
}

void CoordinateMolecularMap::Calculate_No_Wrap_Crd(const VECTOR *crd)
{
    if (is_initialized)
        Calculate_No_Wrap_Crd_CUDA << <blocks_per_grid, threads_per_block >> >(atom_numbers, box_map_times, box_length, crd, nowrap_crd);
}

void CoordinateMolecularMap::Refresh_BoxMapTimes(const VECTOR *crd)
{
    if (is_initialized)
    {
        Refresh_BoxMapTimes_CUDA << <blocks_per_grid, threads_per_block >> >
            (atom_numbers, 1.0 / box_length, crd,
            box_map_times, old_crd);
    }
}

void CoordinateMolecularMap::Update_Volume(VECTOR box_length)
{
    if (!is_initialized)
        return;
    this->box_length = box_length;
}
